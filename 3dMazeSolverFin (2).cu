#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <ctype.h>
#include <sys/time.h>
//#include <time.h>

typedef struct vertex vertex;

struct vertex {
	int _3dCoordinate[3];	//position xyz
	int adjEdges[5][3];
	long long adjIndex[5];
	long index;
	long visited;
	long dist;
	long prev;
};

typedef struct node node;

struct node{
	vertex v;
	node *next;
	node *prev;
};

struct timezone Idunno;	
struct timeval startTime, endTime;

//graph
int entrance[3];
int mazeExit[3];
vertex *vertices;
char *pathToExit;
int degreeOfRandom = 3;

//dijkstra
//long long _INFINITY = 9223372036854775807, totalCount, numEdges = 0;
int _INFINITY = 2147483647, totalCount, numEdges = 0;
node *head;
node *current;

//general
int mazeX = 3, mazeY = 3, mazeZ;
long long numVertices;
int initTmp = 7;

//cuda data
int *mSum;// = &initTmp;	//iterator control for gpu dijkstra
int gpuPathLength;
int numThreads = 1024;


int *E;	//edge array
int *S;	//start position
int *D; 	//degree
int *M;	//mask
int *C;	//cost array
int *U;	//update cost array
int *gpuPath;

int *E_d;
int *S_d;
int *D_d;
int *M_d;
int *C_d;
int *U_d;
int *path_d;
int *mSum_d;

long long *numVertices_d;

void makeGraph(){
	//downCount keeps track of when to add an open floor to the maze
	long long downCount = 0;
	int x = 0, y = 0, z = 0;
	totalCount = 0;
	
	//build maze with known long path to exit with random elements
	//set up all adjacent edges for each vertex
	for(z = 0; z < mazeZ; z++){
		for(y = 0; y < mazeY; y++){
			for(x=0; x < mazeX; x++){
				int a, b;
				
				
				//initlize adjEdges to -1
				for (a = 0; a < 5; a++){
					for(b = 0; b < 3; b++){
						vertices[totalCount].adjEdges[a][b] = -1;
					}
					vertices[totalCount].adjIndex[a] = -1;
				}
				vertices[totalCount].visited = 0;

				vertices[totalCount]._3dCoordinate[0] = x;
				vertices[totalCount]._3dCoordinate[1] = y;
				vertices[totalCount]._3dCoordinate[2] = z;

				//add east adj edge to current vertex
				if(x == 0 && y == 0){
					vertices[totalCount].adjEdges[2][0] = x+1;
					vertices[totalCount].adjEdges[2][1] = y;
					vertices[totalCount].adjEdges[2][2] = z;
					/*
					if(totalCount+1 < numVertices){
						vertices[totalCount].adjIndex[2] = vertices[totalCount+1];
					}
					*/
					
					vertices[totalCount].adjIndex[2] = totalCount+1;

					numEdges++;
					downCount++;
				}
				//add east, west adj edge to current vertex
				else if(x != 0 && x != mazeX-1 && y == 0){
					//east
					vertices[totalCount].adjEdges[2][0] = x+1;
					vertices[totalCount].adjEdges[2][1] = y;
					vertices[totalCount].adjEdges[2][2] = z;
					/*
					if(totalCount+1 < numVertices){
						vertices[totalCount].adjIndex[2] = vertices[totalCount+1];
					}
					*/
					vertices[totalCount].adjIndex[2] = totalCount+1;
					
					//west
					vertices[totalCount].adjEdges[3][0] = x-1;
					vertices[totalCount].adjEdges[3][1] = y;
					vertices[totalCount].adjEdges[3][2] = z;
					/*
					if(totalCount-1 > -1){
						vertices[totalCount].adjIndex[3] = vertices[totalCount-1];
					}
					*/
					vertices[totalCount].adjIndex[3] = totalCount-1;
					numEdges+=2;
				}
				//add north, west adj edge to current vertex
				else if(x == mazeX-1 && y == 0){
					//north
					vertices[totalCount].adjEdges[0][0] = x;
					vertices[totalCount].adjEdges[0][1] = y+1;
					vertices[totalCount].adjEdges[0][2] = z;
					/*
					if (totalCount + mazeX < numVertices){
						vertices[totalCount].adjIndex[0] = vertices[totalCount+mazeX];
					}
					*/
					vertices[totalCount].adjIndex[0] = totalCount+mazeX;
					
					//west
					vertices[totalCount].adjEdges[3][0] = x-1;
					vertices[totalCount].adjEdges[3][1] = y;
					vertices[totalCount].adjEdges[3][2] = z;
					/*
					if(totalCount > -1){
						vertices[totalCount].adjIndex[3] = vertices[totalCount-1];
					}
					*/
					vertices[totalCount].adjIndex[3] = totalCount-1;
					numEdges+=2;
				}
				//add north, south adj edge to current vertex
				else if(x == mazeX-1 && y != mazeY-1 && y != 0){
					//north
					vertices[totalCount].adjEdges[0][0] = x;
					vertices[totalCount].adjEdges[0][1] = y+1;
					vertices[totalCount].adjEdges[0][2] = z;
					/*
					if(totalCount+mazeX < numVertices){
						vertices[totalCount].adjIndex[0] = vertices[totalCount+mazeX];
					}
					*/
					vertices[totalCount].adjIndex[0] = totalCount+mazeX;
					
					//south
					vertices[totalCount].adjEdges[1][0] = x;
					vertices[totalCount].adjEdges[1][1] = y-1;
					vertices[totalCount].adjEdges[1][2] = z;
					/*
					if(totalCount - mazeX > -1){
						vertices[totalCount].adjIndex[1] = vertices[totalCount-mazeX];
					}
					*/
					vertices[totalCount].adjIndex[1] = totalCount-mazeX;
					numEdges+=2;
				}
				//add south adj edge to current vertex
				else if(x == mazeX-1 && y == mazeY-1){
					//south
					vertices[totalCount].adjEdges[1][0] = x;
					vertices[totalCount].adjEdges[1][1] = y-1;
					vertices[totalCount].adjEdges[1][2] = z;
					/*
					if(totalCount-mazeX > -1){
						vertices[totalCount].adjIndex[1] = vertices[totalCount-mazeX];
					}
					*/
					vertices[totalCount].adjIndex[1] = totalCount-mazeX;

					downCount++;
					numEdges++;
				}
				
				//remove floor to every odd plane
				if(x == 0 && y == 0){
					
					if(z % 2 != 0){
						//open floor of this vertex
						vertices[totalCount].adjEdges[4][0] = x;
						vertices[totalCount].adjEdges[4][1] = y;
						vertices[totalCount].adjEdges[4][2] = z+1;
						/*
						if (totalCount + mazeX*mazeY < numVertices){
							vertices[totalCount].adjIndex[4] = vertices[totalCount + mazeX*mazeY];
						}
						*/
						vertices[totalCount].adjIndex[4] = totalCount + mazeX*mazeY;
						numEdges++;
					}
				} 
				
				//remove floor from every even plane
				if (x == mazeX-1 && y == mazeY-1){
					
					if(z % 2 == 0){
						//open floor of this vertex
						vertices[totalCount].adjEdges[4][0] = x;
						vertices[totalCount].adjEdges[4][1] = y;
						vertices[totalCount].adjEdges[4][2] = z+1;
						/*
						if (totalCount + mazeX*mazeY < numVertices){
							vertices[totalCount].adjIndex[4] = vertices[totalCount + mazeX*mazeY];
						}
						*/
						vertices[totalCount].adjIndex[4] = totalCount + mazeX*mazeY;
						numEdges++;
					}
				}
				
				//add random walls/floors------------------------------------------------------------------
				
				int tmprand;
				tmprand = rand() % 10 + 1;
				//rand north
				if(y != mazeY-1 && tmprand <= degreeOfRandom){
					vertices[totalCount].adjIndex[0] = totalCount+mazeX;
					numEdges++;
				}
				tmprand = rand() % 10 + 1;
				//rand south
				if(y != 0 && tmprand <= degreeOfRandom){
					vertices[totalCount].adjIndex[1] = totalCount-mazeX;
					numEdges++;
				}
				tmprand = rand() % 10 + 1;
				//rand east
				if(x != mazeX-1 && tmprand <= degreeOfRandom){
					vertices[totalCount].adjIndex[2] = totalCount + 1;
					numEdges++;
				}
				tmprand = rand() % 10 + 1;
				//rand west
				if(x != 0 && tmprand <= degreeOfRandom){
					vertices[totalCount].adjIndex[3] = totalCount - 1;
					numEdges++;
				}
				tmprand = rand() % 10 + 1;
				//rand floor
				if(z != mazeZ-1 && tmprand<= degreeOfRandom){
					vertices[totalCount].adjIndex[4] = totalCount + mazeX*mazeY;
					numEdges++;
				}
			
				vertices[totalCount].index = totalCount;
				totalCount++;
			}	
		}
	}
	//printAdjIndexes();
}

double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld sec\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

void printAdjIndexes(){
	int i, j;

	for (i = 0; i < numVertices; i++){
		for (j = 0; j < 5; j++){
			printf("%d: %lld\n", i, vertices[i].adjIndex[j]);	
		}
	}
}

void printAdjEdges(){
	int i,k;
	
	for(i=0; i < numVertices; i++){
		for (k = 0; k < 5; k++){
			printf("%d: ", i);
			//for (j = 0; j< 3; j++){
			//	printf("%d", vertices[i].adjEdges[k][j]);
			printf("%d", vertices[i].adjIndex[k]);
			//}
			printf("\n");
		}
	}
	
	printf("total: %d\n", totalCount);
}

//print adjIndexes from linked list
void printLinked(){
	int count = 0;
	node *tmp = head;
	printf("forward traversal\n");
	while(tmp != NULL){
	/*
		for(i=0; i < 5; i++){
			printf("%d: %lld \n", count, tmp->v.adjIndex[i]);
		}
		*/
		printf("%d\n", tmp->v.index);
		count++;
		tmp=tmp->next;
	}
	printf("\n");
}

void printPath(){
	int i;
	i = numVertices-1;
	
	printf("Shortest Path: %d\n", vertices[i].dist);
	while(i != NULL){
		
	
		printf("prev: %d\n", vertices[i].index);
	
		i = vertices[i].prev;
		
	}
}

node * getNewNode(vertex v){

	node * newNode = (node*)malloc(sizeof(node));
	newNode -> v = v;
	newNode -> prev = NULL;
	newNode -> next = NULL;
	return newNode;

}

void insertAtTail(vertex v){

	node * newNode = getNewNode(v);	
		
	if(head == NULL){
		head = newNode;
		current = newNode;
		return;
	}
	
	current->next = newNode;
	newNode->prev = current;
	current = newNode;
}

void popHead(){
	node * tmp = head;

	head = head->next;
	
	if(head != NULL){
		head->prev = NULL;
	}
	free(tmp);
}

void removeNode(node* rem){
	
	node* tmp;
	
	//remove element in middle of list
	if(rem->prev != NULL && rem->next != NULL ){
		//printf("\nmiddle pop: %d\n", rem->v.index);
		//printf("prev: %d	next:%d\n", rem->prev->v.index, rem->next->v.index);
		tmp = rem->prev;
		tmp->next = rem->next;
		
		tmp = rem->next;
		tmp->prev = rem->prev;
		free(rem);
	}
	//remove element at head
	if(rem->prev == NULL && rem->next != NULL){
		//printf("\nhead pop\n");
		popHead();
	}
	//remove element at tail
	if(rem->prev != NULL && rem->next == NULL){
		//pop tail
		//printf("\ntail pop\n");
		tmp = rem->prev;
		tmp->next = NULL;
		free(rem);
	}
	//printLinked();
}

node* getShortestNode(){
	
	node *tmp = head;
	node *shortestNode;
	long long shortestDist = _INFINITY;
	
	//printf("\nget shortest node\n");
	
	while(tmp != NULL){
		//printf("%d: %lld \n", count, tmp->v.adjIndex[i]);
		if(vertices[tmp->v.index].dist < shortestDist){
			shortestDist = vertices[tmp->v.index].dist;
			shortestNode = tmp;
		}
		//count++;
		tmp=tmp->next;
	}
	//printf("shortest Node is: %lld\n", shortestNode->v.index);
	//printf("dist is: %lld", shortestNode->v.dist);
	return shortestNode;
}

void cpuDijkstra(){

	head = NULL;
	current = NULL;
	
	vertices[0].dist = 0;		//source distance
	vertices[0].prev = NULL;	//undefined
	insertAtTail(vertices[0]);
	
	int i;
	long long tmpDist;
	
	//initilize undiscovered list
	for(i=1; i < numVertices; i++){
		vertices[i].dist = _INFINITY;	//pseudo infinity, size of long long
		vertices[i].prev = NULL;	
		
		//add to linked list/undiscovered list
		insertAtTail(vertices[i]);
		
	}
	//printLinked();
	
	node * shortestNode = head;
	//while linked list not empty, do dijkstra
	while (shortestNode->next != NULL){
		shortestNode = getShortestNode();
		
		vertices[shortestNode->v.index].visited = 1;
		tmpDist = vertices[shortestNode->v.index].dist + 1;
		
		
		for(i = 0; i < 5; i++){
			
			//update dist to adjNode if less than tmpDist
			if (	tmpDist < vertices[shortestNode->v.adjIndex[i]].dist && //current dist less than updated dist
				shortestNode->v.adjIndex[i] < numVertices &&		//index is not out of range
				shortestNode->v.adjIndex[i] != -1 &&			//adjIndex is valid
				vertices[shortestNode->v.adjIndex[i]].visited == 0){	//node is unvisited
				
				vertices[shortestNode->v.adjIndex[i]].dist = tmpDist;
				vertices[shortestNode->v.adjIndex[i]].prev = shortestNode->v.index;
			}
		}
		//popHead();
		removeNode(shortestNode);
	}
	//printf("\n");
}

void getEdgeArray(){
	int i, j;
	
	E = (int*) malloc(sizeof(int)*numEdges);
	S = (int*) malloc(sizeof(int)*numVertices);
	D = (int*) malloc(sizeof(int)*numVertices);
	
	for (i = 0; i < numVertices; i++){
		S[i] = -1;
		D[i] = -1;		
	}
	
	int dCount = 0, edgeCount = 0;
	int chk = 0;
	
	for (i = 0; i < numVertices; i++){
		for (j = 0; j < 5; j++){
			if(vertices[i].adjIndex[j] != -1){
			
				if(chk == 0){
					chk = 1;
					S[i] = edgeCount;
				}
				E[edgeCount] = vertices[i].adjIndex[j];
				
				edgeCount++;
				dCount++;
			}
		}
		D[i] = dCount;
		dCount = 0;
		chk = 0;
	}
}

void printEdgeArray(){
	int i = 0;
	//long long int maxN;
	//maxN = sizeof(E)/sizeof(long long int);
	
	printf("\nE array:\n\n");
	
	for(i = 0; i < numEdges; i++){
		printf("%d: %lld \n", i, E[i]);
	}
	
	printf("\nS, D arrays:\n\n");
	
	for(i = 0; i < numVertices; i++){
		printf("%d: s: %d, d: %d\n", i, S[i], D[i]);
	}
}

void copyToGpu(){
	int i;

	//init some data
	mSum = (int*)malloc(sizeof(int));
	mSum = &initTmp;
	
	M = (int*) malloc(sizeof(int)*numVertices);
	C = (int *) malloc(sizeof(int)*numVertices);
	U = (int *) malloc(sizeof(int)*numVertices);
	
	for(i = 0; i < numVertices; i++){
		M[i] = 0;
		C[i] = _INFINITY;
		U[i] = _INFINITY;
	}
	M[0] = 1;
	C[0] = 0;
	U[0] = 0;
	
	numVertices_d = (long long*) malloc(sizeof(long long));

	
	//*numVertices_d = numVertices;
	if(hipMalloc((void**) &numVertices_d, sizeof(long long)) != hipSuccess){
		printf("hipMalloc failed - turkey");
	}
	if(hipMemcpy(numVertices_d, &numVertices, sizeof(long long), hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - turkey");
		hipFree(numVertices_d);
	}

	//make room on gpu
	if(hipMalloc((void**) &E_d, sizeof(int)*numEdges) != hipSuccess){
		printf("hipMalloc failed - E_d");
	}
	if(hipMalloc((void**) &S_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - S_d");
	}
	if(hipMalloc((void**) &D_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - S_d");
	}
	
	if(hipMalloc((void**) &path_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - path_d");
	}
	/*if(hipMalloc((void**) &gpuPath, sizeof(unsigned long long int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - gpuPath");
	}*/
	if(hipMalloc((void**) &M_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - M_d");
	}
	if(hipMalloc((void**) &C_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - C_d");
	}
	if(hipMalloc((void**) &U_d, sizeof(int)*numVertices) != hipSuccess){
		printf("hipMalloc failed - U_d");
	}
	if(hipMalloc((void**) &mSum_d, sizeof(int)) != hipSuccess){
		printf("hipMalloc failed - mSum_d");
	}
		
	//copy to gpu
	if(hipMemcpy(E_d, E, sizeof(int)*numEdges, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - E_d");
		hipFree(E_d);
	}
	if(hipMemcpy(S_d, S, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - S_d");
		hipFree(S_d);
	}
	if(hipMemcpy(D_d, D, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - D_d");
		hipFree(D_d);
	}
	if(hipMemcpy(M_d, M, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - M_d");
		hipFree(M_d);
	}
	if(hipMemcpy(C_d, C, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - C_d");
		hipFree(C_d);
	}
	if(hipMemcpy(U_d, U, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - U_d");
		hipFree(U_d);
	}
	if(hipMemcpy(path_d, gpuPath, sizeof(int)*numVertices, hipMemcpyHostToDevice) != hipSuccess){
		printf("hipMemcpy failed - U_d");
		hipFree(U_d);
	}

}



__global__ void getMsum_kernel(int *M_d, int *mSum_d, long long *numVertices){
	long long i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i ==0){
		*mSum_d = 0;
	}
	__syncthreads();
	
	if(i < *numVertices && M_d[i] > 0){
		atomicAdd((mSum_d), 1);
	}
}

void getMsum(){

	getMsum_kernel<<<(numVertices/256+1), 256>>>(M_d, mSum_d, numVertices_d);
	
		//get mSum from kernel
		if (hipMemcpy(mSum, mSum_d, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
			printf("hipMemcpyDeviceToHost mSum_d failed");
		}
}

__global__ void updateCost_kernel(int * M_d, int *C_d, int *U_d, long long *numVertices){

	long long i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < *numVertices){
		if(C_d[i] > U_d[i]){
			C_d[i] = U_d[i];
			M_d[i] = 1;
		}
		//U_d[i] = C_d[i];
	}
}

__global__ void exploreNeighbors_kernel (int * E_d, int * S_d, int * D_d, int * M_d, int *C_d, int *U_d, int * path_d, long long *numVertices){

	long long i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, neighborPos;
	
	if(i < *numVertices){
		if(M_d[i] == 1){
			M_d[i] = 0;
		
			//for all neighbors of node i
			for (j = 0; j < D_d[i]; j++){
		
				neighborPos = E_d[S_d[i]+j];
			
				//update weight of neighbor if less than current value
				if(U_d[neighborPos] > C_d[i] + 1){
					U_d[neighborPos] = C_d[i] + 1;
					path_d[neighborPos] = i;
				}
			}
		}
		
		__syncthreads();
		
		//update cost
		if(C_d[i] > U_d[i]){
		
			C_d[i] = U_d[i];
			M_d[i] = 1;
		}
	}
}

void getGpuPath(){
	if (hipMemcpy(gpuPath, path_d, sizeof(int)*numVertices, hipMemcpyDeviceToHost) != hipSuccess){
			printf("hipMemcpyDeviceToHost path_d failed");
		}
	//int i;	
	//for(i = 0; i < sizeof(gpuPath)/sizeof(gpuPath[0]){
}

void printGpuPath(){
	//printf("\n\nGpu Path: %d\n\n", sizeof(gpuPath)/sizeof(gpuPath[0]));
	int i = numVertices-1;
	gpuPathLength = 0;
	
	while(i != 0){
		//printf("%d\n", i);
		i = gpuPath[i];
		gpuPathLength++;
	}
	
	/*
	for(i = 0; i < numVertices; i++){
		if(gpuPath[i] != 0){
			printf("%d\n", gpuPath[i]);
		}
	}
	*/
	printf("Gpu Shortest Path: %d\n\n", gpuPathLength);
}

void printGpuFullPath(){
	int i = numVertices-1;
	gpuPathLength = 0;
	
	while(i != 0){
		printf("%d\n", i);
		i = gpuPath[i];
		gpuPathLength++;
	}
	
}

int main(int argc, char **argv){

	srand(time(NULL));

	mazeX = atoi(argv[1]);
	mazeY = atoi(argv[2]);
	mazeZ = atoi(argv[3]);
	degreeOfRandom = atoi(argv[4]);
	numThreads = atoi(argv[5]);
	
	numVertices = mazeX * mazeY * mazeZ;

	gpuPath = (int*) malloc(sizeof(int)*numVertices);	

	mazeExit[0] = mazeX -1;
	mazeExit[1] = mazeY -1;
	mazeExit[2] = mazeZ -1;

	vertices = (vertex *)malloc(sizeof(vertex)*numVertices);

	makeGraph();
	
	printf("\nNumver of Vertices: %lld", numVertices);
	printf("\nNumber of Edges: %lld\n\n", numEdges);
	
	/* start counting time */
	gettimeofday(&startTime, &Idunno);
	
	cpuDijkstra();
	
	/* check the total running time */ 
	report_running_time();
	
	printf("CPU Shortest Path: %d\n\n", vertices[numVertices-1].dist);
	//printPath();	//cpu
//---------------------------------------------------------------------------------------------------
	getEdgeArray();			//gets single array that holds all edges E for the graph
					//also makes start array S and degree array D
	//printEdgeArray();
	
	copyToGpu();
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);	
	
	//Do cuda dijkstra
	while(*mSum != 0){
		
		exploreNeighbors_kernel<<<(numVertices/numThreads+1), numThreads>>>(E_d, S_d, D_d, M_d, C_d, U_d, path_d, numVertices_d);
		//updateCost_kernel<<<(numVertices/numThreads+1), numThreads>>>(M_d, C_d, U_d, numVertices_d);
	
		if (hipMemcpy(mSum, mSum_d, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
			//hipFree(mSum_d, numVertices);
			printf("hipMemcpyDeviceToHost mSum_d failed");
		}
	
		//printf("mSum: %d\n", *mSum);
		getMsum();
		//printf("mSum: %d\n", *mSum);
		
		//printf("\n%d: mSum: %d\n",loopCounter++, *mSum);
		
	}
	
	getGpuPath();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Gpu time: %0.6f sec\n", elapsedTime/1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	//printf("\n\nGpu Path: %d\n\n", (unsigned long long int) sizeof(gpuPath));
	printGpuPath();
	//printGpuFullPath();
	
	return 0;
}
